#include "hip/hip_runtime.h"
#include <GL/glut.h>
#include <thrust/complex.h>

//#include <hip/hip_runtime.h>
//#include <cuda_gl_interop.h>

#include "fractal.h"

#define PI 3.1415926535

using namespace thrust;

extern "C"
void compute(GLubyte* data, const int width, const int height, const int iterations,
             const double midx, const double midy, const double scale,
             const double varx, const double vary, const bool julia,
             RenderType type, GLubyte* colorSpectrum, const int colorSpectrumSize,
             const GLubyte backr, const GLubyte backg, const GLubyte backb);

__global__ void computeEscape(GLubyte* data, const int imgWidth, const int imgHeight, const int iterations,
                              const double midx, const double midy, const double scale,
                              const double varx, const double vary, const bool julia,
                              GLubyte* colorSpectrum, const int colorSpectrumSize,
                              const GLubyte backr, const GLubyte backg, const GLubyte backb)
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = index_y * imgWidth + index_x;


    double ax, ay;
    if (imgWidth > imgHeight) {
        ax = (double)imgWidth / imgHeight;
        ay = 1.0f;
    } else {
        ax = 1.0f;
        ay = (double)imgHeight / imgWidth;
    }

    if (index_x < imgWidth && index_y < imgHeight) {
        double a, b, x, y, asq, bsq, rsq, e, atemp, nor;

        a = midx + 2.0 * ax * scale * (double)(2.0 * index_x - imgWidth) / imgWidth;
        b = midy + 2.0 * ay * scale * (double)(2.0 * index_y - imgHeight) / imgHeight;

        if (julia) {
            x = varx;
            y = vary;
        } else {
            x = a + varx;
            y = b + vary;
        }

        complex<double> z = complex<double>(a, b);
        complex<double> c = complex<double>(x, y);

        rsq = 4;
        nor = norm(z);
        int k = 0;

        while (nor < rsq && k < iterations) {
            //f(z) = z^2 + c (mandelbrot)
            z = z*z + c;

            e += expf(-nor);
            ++k;
            nor = norm(z);
        }

        /*
        asq = a * a;
        bsq = b * b;
        rsq = 4;
        e = 0;


        int k = 0;

        //fractal formulas
        //http://www.lifesmith.com/formulas.html
        while (asq + bsq < rsq && k < iterations) {

            //f(z) = z^2 + c (mandelbrot)
            atemp = asq - bsq + x;
            b = a * b;
            b += b + y;
            a = atemp;

            //f(z) = 1/z + c
            //double m = asq + bsq;
            //atemp = a / m + x;
            //b = -b / m + y;
            //a = atemp;

            //f(z) = z^2 + 1 / z + c + d
            //where d = e^(2pi/3*i) * 3 / 2^(2/3)
            //double m = asq + bsq;
            //atemp = asq - bsq + a / m + x + 0.944940787421154874;
            //b     = 2 * a * b - b / m + y - 1.636685453957582105;
            //a = atemp;

            //f(z) = (|Re(z)| + i|Im(z)|)^2 + c (burning ship)
            //atemp = asq - bsq + x;
            //b = -abs(a * b); //turn upside down
            //b += b + y;
            //a = atemp;

            //f(a + bi) = (|a| + |b|i) / |a + bi| + c
            //double m = asq + bsq;
            //a = abs(a) / m + x;
            //b = abs(b) / m + y;

            //f(z) = 1/z^2 + c
            //double m = asq + bsq;
            //m *= m;
            //atemp = (asq - bsq) / m + x;
            //b = -(2 * a * b) / m + y;
            //a = atemp;

            //f(z) = z^4 + c/z
            //double m = asq + bsq;
            //atemp = asq * asq - 6 * asq * bsq + bsq * bsq + (a * x + b * y) / m;
            //b = 4 * a * b * (asq - bsq) + (a * y - b * x) / m;
            //a = atemp;

            //f(z) = z^2 + c/z^3
            //double u = a * (asq - 3 * bsq);
            //double v = b * (3 * asq - bsq);
            //double m = u * u + v * v;
            //atemp = asq - bsq + (u * x + v * y) / m;
            //b = 2 * a * b + (u * y - v * x) / m;
            //a = atemp;

            //f(z) = z^4 + c/z^3
            //double u = a * (asq - 3 * bsq);
            //double v = b * (3 * asq - bsq);
            //double m = u * u + v * v;
            //atemp = asq * asq - 6 * asq * bsq + bsq * bsq + (u * x + v * y) / m;
            //b = 4 * a * b * (asq - bsq) + (u * y - v * x) / m;
            //a = atemp;

            //f(z) = z^4 + c/z^4
            //double u = asq * asq - 6 * asq * bsq + bsq * bsq;
            //double v = 4 * a * b * (asq - bsq);
            //double m = u * u + v * v;
            //atemp = u + (u * x + v * y) / m;
            //b = v + (u * y - v * x) / m;
            //a = atemp;

            e += expf(-(asq + bsq)); //do not use sqrt as it does not add much

            asq = a * a;
            bsq = b * b;
            ++k;
        }*/

        int j = 4 * i;

        if (k == iterations) {
            data[j] = backr;
            data[j + 1] = backg;
            data[j + 2] = backb;
            data[j + 3] = 255;
        } else {
            float hue = (0.025f * e - (int)(0.025f * e));

            //float f = atan(z.imag()/z.real()) / PI + 0.5;
            //float huetemp = hue + f;
            //hue = huetemp - (int)huetemp;

            //float f = norm(z);
            //float huetemp = hue + f;
            //hue = huetemp - (int)huetemp;

            int n = (int)(hue * (colorSpectrumSize - 1));
            float h = hue * (colorSpectrumSize - 1) - n;

            GLubyte r1 = colorSpectrum[3 * n];
            GLubyte g1 = colorSpectrum[3 * n + 1];
            GLubyte b1 = colorSpectrum[3 * n + 2];
            GLubyte r2 = colorSpectrum[3 * n + 3];
            GLubyte g2 = colorSpectrum[3 * n + 4];
            GLubyte b2 = colorSpectrum[3 * n + 5];

            GLubyte R, G, B;
            R = r1 * (1 - h) + r2 * h;
            G = g1 * (1 - h) + g2 * h;
            B = b1 * (1 - h) + b2 * h;

            data[j] = R;
            data[j + 1] = G;
            data[j + 2] = B;
            data[j + 3] = 255;
        }
    }
}

/*__global__ void computeNewton(GLubyte* data, const int imgWidth, const int imgHeight, const int iterations,
                              const double midx, const double midy, const double scale,
                              const double varx, const double vary, const bool julia,
                              GLubyte* colorSpectrum, const int colorSpectrumSize)
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = index_y * imgWidth + index_x;


    double ax, ay;
    if (imgWidth > imgHeight) {
        ax = (double)imgWidth / imgHeight;
        ay = 1.0f;
    } else {
        ax = 1.0f;
        ay = (double)imgHeight / imgWidth;
    }

    if (index_x < imgWidth && index_y < imgHeight) {
        double a, b, x, y, asq, bsq, epsilon, e, atemp;

        a = midx + 2.0 * ax * scale * (double)(2.0 * index_x - imgWidth) / imgWidth;
        b = midy + 2.0 * ay * scale * (double)(2.0 * index_y - imgHeight) / imgHeight;

        if (julia) {
            x = varx;
            y = vary;
        } else {
            x = a + varx;
            y = b + vary;
        }

        asq = a * a;
        bsq = b * b;
        e = 0;

        epsilon = 1E-10;
        double r1 = 1;
        double s1 = 0;
        double r2 = -0.5;
        double s2 = sqrtf(3) / 2;
        double r3 = -0.5;
        double s3 = -sqrtf(3) / 2;

        double l1, l2, l3;
        l1 = (a-r1)*(a-r1)+(b-s1)*(b-s1);
        l2 = (a-r2)*(a-r2)+(b-s2)*(b-s2);
        l3 = (a-r3)*(a-r3)+(b-s3)*(b-s3);

        int k = 0;

        while (l1 > epsilon && l2 > epsilon && l3 > epsilon && k < iterations) {

            //f(z) = z - (1+c)p(z) / p'(z)
            //p(z) = z^3 - 1
            double u1 = a * asq - 3 * a * bsq - 1;
            double v1 = 3 * asq * b - bsq * b;
            double u2 = asq - bsq;
            double v2 = 2 * a * b;
            double m = 3 * (u2 * u2 + v2 * v2);
            double u3 = (u1 * u2 + v1 * v2) / m;
            double v3 = (u2 * v1 - u1 * v2) / m;

            atemp = a - (1+x)*u3 + y*v3;
            b = b - (1+x)*v3 - y*u3;
            a = atemp;

            e += expf(-1/l1-1/l2-1/l3);

            asq = a * a;
            bsq = b * b;
            ++k;

            l1 = (a-r1)*(a-r1)+(b-s1)*(b-s1);
            l2 = (a-r2)*(a-r2)+(b-s2)*(b-s2);
            l3 = (a-r3)*(a-r3)+(b-s3)*(b-s3);
        }

        int j = 4 * i;

        if (k == iterations) {
            data[j] = 0;
            data[j + 1] = 0;
            data[j + 2] = 0;
            data[j + 3] = 255;
        } else {
            float hue = (0.025f * e - (int)(0.025f * e));

            if ((a-r2)*(a-r2)+(b-s2)*(b-s2) < epsilon) {
                double huetemp = hue + 1.0/3;
                hue = huetemp - (int)huetemp;
            } else if ((a-r3)*(a-r3)+(b-s3)*(b-s3) < epsilon) {
                double huetemp = hue + 2.0/3;
                hue = huetemp - (int)huetemp;
            }

            int n = (int)(hue * (colorSpectrumSize - 1));
            float h = hue * (colorSpectrumSize - 1) - n;

            GLubyte r1 = colorSpectrum[3 * n];
            GLubyte g1 = colorSpectrum[3 * n + 1];
            GLubyte b1 = colorSpectrum[3 * n + 2];
            GLubyte r2 = colorSpectrum[3 * n + 3];
            GLubyte g2 = colorSpectrum[3 * n + 4];
            GLubyte b2 = colorSpectrum[3 * n + 5];

            GLubyte R, G, B;
            R = r1 * (1 - h) + r2 * h;
            G = g1 * (1 - h) + g2 * h;
            B = b1 * (1 - h) + b2 * h;

            data[j] = R;
            data[j + 1] = G;
            data[j + 2] = B;
            data[j + 3] = 255;
        }
    }
}*/


__global__ void computeNewton(GLubyte* data, const int imgWidth, const int imgHeight, const int iterations,
                              const double midx, const double midy, const double scale,
                              const double varx, const double vary, const bool julia,
                              GLubyte* colorSpectrum, const int colorSpectrumSize,
                              const GLubyte backr, const GLubyte backg, const GLubyte backb)
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = index_y * imgWidth + index_x;


    double ax, ay;
    if (imgWidth > imgHeight) {
        ax = (double)imgWidth / imgHeight;
        ay = 1.0f;
    } else {
        ax = 1.0f;
        ay = (double)imgHeight / imgWidth;
    }

    if (index_x < imgWidth && index_y < imgHeight) {
        double a, b, x, y, asq, bsq, epsilon, e, l;

        a = midx + 2.0 * ax * scale * (double)(2.0 * index_x - imgWidth) / imgWidth;
        b = midy + 2.0 * ay * scale * (double)(2.0 * index_y - imgHeight) / imgHeight;

        if (julia) {
            x = varx;
            y = vary;
        } else {
            x = a + varx;
            y = b + vary;
        }

        complex<double> z = complex<double>(a, b);
        complex<double> c = complex<double>(x, y);
        complex<double> z1;

        e = 0;
        epsilon = 1E-5;
        l = 1;
        int k = 0;

        while (l > epsilon && k < iterations) {
            //old value
            z1 = z;

            //f(z) = z - (1+c)p(z) / p'(z)
            //p(z) = z^3 - 1
            //z = z - ((1.0+c) * (z*z*z - 1.0) / (3.0*z*z));

            //f(z) = z - (1+c)p(z) / p'(z)
            //p(z) = z^6 + z^3 - 1
            //z = z - ((1.0+c) * (z*z*z*z*z*z + z*z*z - 1.0) / (6.0*z*z*z*z*z + 3.0*z*z));

            z = z - ((1.0+c) * sin(z) / cos(z));

            l = norm(z - z1);
            e += expf(-1/l);

            ++k;
        }

        int j = 4 * i;

        if (k == iterations) {
            data[j] = backr;
            data[j + 1] = backg;
            data[j + 2] = backb;
            data[j + 3] = 255;
        } else {
            float hue = (0.025f * e - (int)(0.025f * e));

            //float f = atan(z.imag()/z.real()) / PI + 0.5;
            //float huetemp = hue + f;
            //hue = huetemp - (int)huetemp;

            float f = norm(z);
            float huetemp = hue + f;
            hue = huetemp - (int)huetemp;

            int n = (int)(hue * (colorSpectrumSize - 1));
            float h = hue * (colorSpectrumSize - 1) - n;

            GLubyte r1 = colorSpectrum[3 * n];
            GLubyte g1 = colorSpectrum[3 * n + 1];
            GLubyte b1 = colorSpectrum[3 * n + 2];
            GLubyte r2 = colorSpectrum[3 * n + 3];
            GLubyte g2 = colorSpectrum[3 * n + 4];
            GLubyte b2 = colorSpectrum[3 * n + 5];

            GLubyte R, G, B;
            R = r1 * (1 - h) + r2 * h;
            G = g1 * (1 - h) + g2 * h;
            B = b1 * (1 - h) + b2 * h;

            data[j] = R;
            data[j + 1] = G;
            data[j + 2] = B;
            data[j + 3] = 255;
        }
    }
}


__global__



__global__ void computeOrbit(GLubyte* data, const int imgWidth, const int imgHeight, const int iterations,
                             const double midx, const double midy, const double scale,
                             const double varx, const double vary, const bool julia,
                             const double pointx, const double pointy,
                             GLubyte* colorSpectrum, const int colorSpectrumSize)
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = index_y * imgWidth + index_x;


    double ax, ay;
    if (imgWidth > imgHeight) {
        ax = (double)imgWidth / imgHeight;
        ay = 1.0f;
    } else {
        ax = 1.0f;
        ay = (double)imgHeight / imgWidth;
    }

    if (index_x < imgWidth && index_y < imgHeight) {
        double a, b, x, y, asq, bsq, rsq, atemp, distance, xm, ym, newDistance;

        a = midx + 2.0 * ax * scale * (double)(2.0 * index_x - imgWidth) / imgWidth;
        b = midy + 2.0 * ay * scale * (double)(2.0 * index_y - imgHeight) / imgHeight;

        if (julia) {
            //x = varx;
            //y = vary;
            x = 0.5;
            y = 0.3;
        } else {
            x = a + varx;
            y = b + vary;
        }

        asq = a * a;
        bsq = b * b;
        rsq = 1 << 20;
        distance = 1 << 20;


        int k = 0;

        //fractal formulas
        //http://www.lifesmith.com/formulas.html
        while (asq + bsq < rsq && k < iterations) {

            //f(z) = z^2 + c (mandelbrot)
            atemp = asq - bsq + x;
            b = a * b;
            b += b + y;
            a = atemp;

            //f(z) = z^2 + 1 / z + c + d
            //where d = e^(2pi/3*i) * 3 / 2^(2/3)
            //double m = asq + bsq;
            //atemp = asq - bsq + a / m + x + 0.944940787421154874;
            //b     = 2 * a * b - b / m + y - 1.636685453957582105;
            //a = atemp;

            //f(z) = (|Re(z)| + i|Im(z)|)^2 + c (burning ship)
            //atemp = asq - bsq + x;
            //b = abs(a * b);
            //b += b + y;
            //a = atemp;

            //f(a + bi) = (|a| + |b|i) / |a + bi| + c
            //double m = asq + bsq;
            //a = abs(a) / m + x;
            //b = abs(b) / m + y;

            //f(z) = 1/z^2 + c
            //double m = asq + bsq;
            //m *= m;
            //atemp = (asq - bsq) / m + x;
            //b = -(2 * a * b) / m + y;
            //a = atemp;


            xm = a - pointx;
            ym = b - pointy;

            newDistance = xm * xm + ym * ym;
            //newDistance = abs(xm) + abs(ym);
            //newDistance = abs(9 - (xm * xm + ym * ym));
            //newDistance = abs((xm + ym) / 2);

            if (distance > newDistance)
                distance = newDistance;


            asq = a * a;
            bsq = b * b;
            ++k;
        }

        float hue = (0.1f * distance - (int)(0.1f * distance));

        int n = (int)(hue * (colorSpectrumSize - 1));
        float h = hue * (colorSpectrumSize - 1) - n;

        GLubyte r1 = colorSpectrum[3 * n];
        GLubyte g1 = colorSpectrum[3 * n + 1];
        GLubyte b1 = colorSpectrum[3 * n + 2];
        GLubyte r2 = colorSpectrum[3 * n + 3];
        GLubyte g2 = colorSpectrum[3 * n + 4];
        GLubyte b2 = colorSpectrum[3 * n + 5];

        GLubyte R, G, B;
        R = r1 * (1 - h) + r2 * h;
        G = g1 * (1 - h) + g2 * h;
        B = b1 * (1 - h) + b2 * h;

        int j = 4 * i;
        data[j] = R;
        data[j + 1] = G;
        data[j + 2] = B;
        data[j + 3] = 255;
    }
}


__global__ void computeAverage(GLubyte* data, const int imgWidth, const int imgHeight, const int iterations,
                               const double midx, const double midy, const double scale,
                               const double varx, const double vary, const bool julia,
                               GLubyte* colorSpectrum, const int colorSpectrumSize)
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = index_y * imgWidth + index_x;


    double ax, ay;
    if (imgWidth > imgHeight) {
        ax = (double)imgWidth / imgHeight;
        ay = 1.0f;
    } else {
        ax = 1.0f;
        ay = (double)imgHeight / imgWidth;
    }

    if (index_x < imgWidth && index_y < imgHeight) {
        double a, b, x, y, asq, bsq, rsq, xsq, ysq, e, atemp, mn, Mn;

        a = midx + 2.0 * ax * scale * (double)(2.0 * index_x - imgWidth) / imgWidth;
        b = midy + 2.0 * ay * scale * (double)(2.0 * index_y - imgHeight) / imgHeight;

        if (julia) {
            x = varx;
            y = vary;
        } else {
            x = a + varx;
            y = b + vary;
        }

        asq = a * a;
        bsq = b * b;
        xsq = x * x;
        ysq = y * y;
        rsq = 1 << 16;
        e = 0;


        int k = 0;

        //fractal formulas
        //http://www.lifesmith.com/formulas.html
        while (asq + bsq < rsq && k < iterations) {

            mn = abs(asq + bsq - xsq - ysq);
            Mn = asq + bsq + xsq + ysq;

            //f(z) = z^2 + c (mandelbrot)
            atemp = asq - bsq + x;
            b = a * b;
            b += b + y;
            a = atemp;

            //f(z) = z^2 + 1 / z + c + d
            //where d = e^(2pi/3*i) * 3 / 2^(2/3)
            //double m = asq + bsq;
            //atemp = asq - bsq + a / m + x + 0.944940787421154874;
            //b     = 2 * a * b - b / m + y - 1.636685453957582105;
            //a = atemp;

            //f(z) = (|Re(z)| + i|Im(z)|)^2 + c (burning ship)
            //atemp = asq - bsq + x;
            //b = abs(a * b);
            //b += b + y;
            //a = atemp;

            //f(z) = (|Re(z)| + i|Im(z)|) / |z| + c
            //double m = asq + bsq;
            //a = abs(a) / m + x;
            //b = abs(b) / m + y;

            //f(z) = 1/z^2 + c
            //double m = asq + bsq;
            //m *= m;
            //atemp = (asq - bsq) / m + x;
            //b = -(2 * a * b) / m + y;
            //a = atemp;

            e += 0.5 + 0.5 * sin(2.0 * atan2(b, a));
            //e += (asq + bsq - mn) / (Mn - mn);

            asq = a * a;
            bsq = b * b;
            ++k;
        }

        double lastOrbit = 0.5 + 0.5 * sin(2.0 * atan2(b, a));
        //double lastOrbit = (asq + bsq - mn) / (Mn - mn);

        double f = e - lastOrbit;
        e /= k;
        f /= k - 1;
        float frac = 1.0 + log(log(rsq) / log(asq + bsq)) / log(2.0);
        float mix = frac * e + (1 - frac) * f;
        float hue;

        //hue = mix;


        if (k == iterations) {
            hue = 1.0;
        } else if (mix >= 0) {
            hue = 1.0f * mix - (int)(1.0f * mix);
        } else {
            hue = 1.0f * mix - (int)(1.0f * mix) + 1;
        }

        int n = (int)(hue * (colorSpectrumSize - 1));
        float h = hue * (colorSpectrumSize - 1) - n;

        GLubyte r1 = colorSpectrum[3 * n];
        GLubyte g1 = colorSpectrum[3 * n + 1];
        GLubyte b1 = colorSpectrum[3 * n + 2];
        GLubyte r2 = colorSpectrum[3 * n + 3];
        GLubyte g2 = colorSpectrum[3 * n + 4];
        GLubyte b2 = colorSpectrum[3 * n + 5];

        GLubyte R, G, B;
        R = r1 * (1 - h) + r2 * h;
        G = g1 * (1 - h) + g2 * h;
        B = b1 * (1 - h) + b2 * h;

        int j = 4 * i;
        data[j] = R;
        data[j + 1] = G;
        data[j + 2] = B;
        data[j + 3] = 255;
    }
}

void compute(GLubyte* data, const int width, const int height, const int iterations,
             const double midx, const double midy, const double scale,
             const double varx, const double vary, const bool julia,
             RenderType renderType, GLubyte* colorSpectrum, const int colorSpectrumSize,
             const GLubyte backr, const GLubyte backg, const GLubyte backb)
{
    dim3 blockSize;
    blockSize.x = 16;
    blockSize.y = 16;

    dim3 gridSize;
    gridSize.x = width / blockSize.x + 1;
    gridSize.y = height / blockSize.y + 1;

    switch (renderType) {
        case Esc:
            computeEscape <<< gridSize, blockSize >>> (data, width, height, iterations, midx, midy, scale, varx, vary, julia, colorSpectrum, colorSpectrumSize, backr, backg, backb);
            break;
        case Orbit:
            computeOrbit <<< gridSize, blockSize >>> (data, width, height, iterations, midx, midy, scale, 0, 0, julia, varx, vary, colorSpectrum, colorSpectrumSize);
            break;
        case Average:
            computeAverage <<< gridSize, blockSize >>> (data, width, height, iterations, midx, midy, scale, varx, vary, julia, colorSpectrum, colorSpectrumSize);
            break;
        case Newton:
            computeNewton <<< gridSize, blockSize >>> (data, width, height, iterations, midx, midy, scale, varx, vary, julia, colorSpectrum, colorSpectrumSize, backr, backg, backb);
            break;
    }
}
