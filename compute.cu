#include "hip/hip_runtime.h"
#include <GL/glut.h>

//#include <hip/hip_runtime.h>
//#include <cuda_gl_interop.h>

#include "fractal.h"

extern "C"
void compute(GLubyte* data, const int width, const int height, const int iterations,
             const double midx, const double midy, const double scale,
             const double varx, const double vary, const bool julia,
             RenderType type, GLubyte* colorSpectrum, const int colorSpectrumSize);

__global__ void computeEscape(GLubyte* data, const int imgWidth, const int imgHeight, const int iterations,
                              const double midx, const double midy, const double scale,
                              const double varx, const double vary, const bool julia,
                              GLubyte* colorSpectrum, const int colorSpectrumSize)
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = index_y * imgWidth + index_x;


    double ax, ay;
    if (imgWidth > imgHeight) {
        ax = (double)imgWidth / imgHeight;
        ay = 1.0f;
    } else {
        ax = 1.0f;
        ay = (double)imgHeight / imgWidth;
    }

    if (index_x < imgWidth && index_y < imgHeight) {
        double a, b, x, y, asq, bsq, rsq, e, atemp;

        a = midx + 2.0 * ax * scale * (double)(2.0 * index_x - imgWidth) / imgWidth;
        b = midy + 2.0 * ay * scale * (double)(2.0 * index_y - imgHeight) / imgHeight;

        if (julia) {
            x = varx;
            y = vary;
        } else {
            x = a + varx;
            y = b + vary;
        }

        asq = a * a;
        bsq = b * b;
        rsq = 4;
        e = 0;


        int k = 0;

        //fractal formulas
        //http://www.lifesmith.com/formulas.html
        while (asq + bsq < rsq && k < iterations) {

            //f(z) = z^2 + c (mandelbrot)
            atemp = asq - bsq + x;
            b = a * b;
            b += b + y;
            a = atemp;

            //f(z) = z^2 + 1 / z + c + d
            //where d = e^(2pi/3*i) * 3 / 2^(2/3)
            //double m = asq + bsq;
            //atemp = asq - bsq + a / m + x + 0.944940787421154874;
            //b     = 2 * a * b - b / m + y - 1.636685453957582105;
            //a = atemp;

            //f(z) = (|Re(z)| + i|Im(z)|)^2 + c (burning ship)
            //atemp = asq - bsq + x;
            //b = -abs(a * b); //turn upside down
            //b += b + y;
            //a = atemp;

            //f(a + bi) = (|a| + |b|i) / |a + bi| + c
            //double m = asq + bsq;
            //a = abs(a) / m + x;
            //b = abs(b) / m + y;

            //f(z) = 1/z^2 + c
            //double m = asq + bsq;
            //m *= m;
            //atemp = (asq - bsq) / m + x;
            //b = -(2 * a * b) / m + y;
            //a = atemp;

            //f(z) = z^4 + c/z
            //double m = asq + bsq;
            //atemp = asq * asq - 6 * asq * bsq + bsq * bsq + (a * x + b * y) / m;
            //b = 4 * a * b * (asq - bsq) + (a * y - b * x) / m;
            //a = atemp;

            //f(z) = z^2 + c/z^3
            //double u = a * (asq - 3 * bsq);
            //double v = b * (3 * asq - bsq);
            //double m = u * u + v * v;
            //atemp = asq - bsq + (u * x + v * y) / m;
            //b = 2 * a * b + (u * y - v * x) / m;
            //a = atemp;

            //f(z) = z^4 + c/z^3
            //double u = a * (asq - 3 * bsq);
            //double v = b * (3 * asq - bsq);
            //double m = u * u + v * v;
            //atemp = asq * asq - 6 * asq * bsq + bsq * bsq + (u * x + v * y) / m;
            //b = 4 * a * b * (asq - bsq) + (u * y - v * x) / m;
            //a = atemp;

            //f(z) = z^4 + c/z^4
            //double u = asq * asq - 6 * asq * bsq + bsq * bsq;
            //double v = 4 * a * b * (asq - bsq);
            //double m = u * u + v * v;
            //atemp = u + (u * x + v * y) / m;
            //b = v + (u * y - v * x) / m;
            //a = atemp;


            e += expf(-(asq + bsq)); //do not use sqrt as it does not add much

            asq = a * a;
            bsq = b * b;
            ++k;
        }

        int j = 4 * i;

        if (k == iterations) {
            data[j] = 0;
            data[j + 1] = 0;
            data[j + 2] = 0;
            data[j + 3] = 255;
        } else {
            float hue = (0.025f * e - (int)(0.025f * e));
            int n = (int)(hue * (colorSpectrumSize - 1));
            float h = hue * (colorSpectrumSize - 1) - n;

            GLubyte r1 = colorSpectrum[3 * n];
            GLubyte g1 = colorSpectrum[3 * n + 1];
            GLubyte b1 = colorSpectrum[3 * n + 2];
            GLubyte r2 = colorSpectrum[3 * n + 3];
            GLubyte g2 = colorSpectrum[3 * n + 4];
            GLubyte b2 = colorSpectrum[3 * n + 5];

            GLubyte R, G, B;
            R = r1 * (1 - h) + r2 * h;
            G = g1 * (1 - h) + g2 * h;
            B = b1 * (1 - h) + b2 * h;

            data[j] = R;
            data[j + 1] = G;
            data[j + 2] = B;
            data[j + 3] = 255;
        }
    }
}


__global__ void computeOrbit(GLubyte* data, const int imgWidth, const int imgHeight, const int iterations,
                             const double midx, const double midy, const double scale,
                             const double varx, const double vary, const bool julia,
                             const double pointx, const double pointy,
                             GLubyte* colorSpectrum, const int colorSpectrumSize)
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = index_y * imgWidth + index_x;


    double ax, ay;
    if (imgWidth > imgHeight) {
        ax = (double)imgWidth / imgHeight;
        ay = 1.0f;
    } else {
        ax = 1.0f;
        ay = (double)imgHeight / imgWidth;
    }

    if (index_x < imgWidth && index_y < imgHeight) {
        double a, b, x, y, asq, bsq, rsq, atemp, distance, xm, ym, newDistance;

        a = midx + 2.0 * ax * scale * (double)(2.0 * index_x - imgWidth) / imgWidth;
        b = midy + 2.0 * ay * scale * (double)(2.0 * index_y - imgHeight) / imgHeight;

        if (julia) {
            //x = varx;
            //y = vary;
            x = 0.5;
            y = 0.3;
        } else {
            x = a + varx;
            y = b + vary;
        }

        asq = a * a;
        bsq = b * b;
        rsq = 1 << 20;
        distance = 1 << 20;


        int k = 0;

        //fractal formulas
        //http://www.lifesmith.com/formulas.html
        while (asq + bsq < rsq && k < iterations) {

            //f(z) = z^2 + c (mandelbrot)
            atemp = asq - bsq + x;
            b = a * b;
            b += b + y;
            a = atemp;

            //f(z) = z^2 + 1 / z + c + d
            //where d = e^(2pi/3*i) * 3 / 2^(2/3)
            //double m = asq + bsq;
            //atemp = asq - bsq + a / m + x + 0.944940787421154874;
            //b     = 2 * a * b - b / m + y - 1.636685453957582105;
            //a = atemp;

            //f(z) = (|Re(z)| + i|Im(z)|)^2 + c (burning ship)
            //atemp = asq - bsq + x;
            //b = abs(a * b);
            //b += b + y;
            //a = atemp;

            //f(a + bi) = (|a| + |b|i) / |a + bi| + c
            //double m = asq + bsq;
            //a = abs(a) / m + x;
            //b = abs(b) / m + y;

            //f(z) = 1/z^2 + c
            //double m = asq + bsq;
            //m *= m;
            //atemp = (asq - bsq) / m + x;
            //b = -(2 * a * b) / m + y;
            //a = atemp;


            xm = a - pointx;
            ym = b - pointy;

            newDistance = xm * xm + ym * ym;
            //newDistance = abs(xm) + abs(ym);
            //newDistance = abs(9 - (xm * xm + ym * ym));
            //newDistance = abs((xm + ym) / 2);

            if (distance > newDistance)
                distance = newDistance;


            asq = a * a;
            bsq = b * b;
            ++k;
        }

        float hue = (0.1f * distance - (int)(0.1f * distance));

        int n = (int)(hue * (colorSpectrumSize - 1));
        float h = hue * (colorSpectrumSize - 1) - n;

        GLubyte r1 = colorSpectrum[3 * n];
        GLubyte g1 = colorSpectrum[3 * n + 1];
        GLubyte b1 = colorSpectrum[3 * n + 2];
        GLubyte r2 = colorSpectrum[3 * n + 3];
        GLubyte g2 = colorSpectrum[3 * n + 4];
        GLubyte b2 = colorSpectrum[3 * n + 5];

        GLubyte R, G, B;
        R = r1 * (1 - h) + r2 * h;
        G = g1 * (1 - h) + g2 * h;
        B = b1 * (1 - h) + b2 * h;

        int j = 4 * i;
        data[j] = R;
        data[j + 1] = G;
        data[j + 2] = B;
        data[j + 3] = 255;
    }
}


__global__ void computeAverage(GLubyte* data, const int imgWidth, const int imgHeight, const int iterations,
                               const double midx, const double midy, const double scale,
                               const double varx, const double vary, const bool julia,
                               GLubyte* colorSpectrum, const int colorSpectrumSize)
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = index_y * imgWidth + index_x;


    double ax, ay;
    if (imgWidth > imgHeight) {
        ax = (double)imgWidth / imgHeight;
        ay = 1.0f;
    } else {
        ax = 1.0f;
        ay = (double)imgHeight / imgWidth;
    }

    if (index_x < imgWidth && index_y < imgHeight) {
        double a, b, x, y, asq, bsq, rsq, xsq, ysq, e, atemp, mn, Mn;

        a = midx + 2.0 * ax * scale * (double)(2.0 * index_x - imgWidth) / imgWidth;
        b = midy + 2.0 * ay * scale * (double)(2.0 * index_y - imgHeight) / imgHeight;

        if (julia) {
            x = varx;
            y = vary;
        } else {
            x = a + varx;
            y = b + vary;
        }

        asq = a * a;
        bsq = b * b;
        xsq = x * x;
        ysq = y * y;
        rsq = 1 << 16;
        e = 0;


        int k = 0;

        //fractal formulas
        //http://www.lifesmith.com/formulas.html
        while (asq + bsq < rsq && k < iterations) {

            mn = abs(asq + bsq - xsq - ysq);
            Mn = asq + bsq + xsq + ysq;

            //f(z) = z^2 + c (mandelbrot)
            atemp = asq - bsq + x;
            b = a * b;
            b += b + y;
            a = atemp;

            //f(z) = z^2 + 1 / z + c + d
            //where d = e^(2pi/3*i) * 3 / 2^(2/3)
            //double m = asq + bsq;
            //atemp = asq - bsq + a / m + x + 0.944940787421154874;
            //b     = 2 * a * b - b / m + y - 1.636685453957582105;
            //a = atemp;

            //f(z) = (|Re(z)| + i|Im(z)|)^2 + c (burning ship)
            //atemp = asq - bsq + x;
            //b = abs(a * b);
            //b += b + y;
            //a = atemp;

            //f(z) = (|Re(z)| + i|Im(z)|) / |z| + c
            //double m = asq + bsq;
            //a = abs(a) / m + x;
            //b = abs(b) / m + y;

            //f(z) = 1/z^2 + c
            //double m = asq + bsq;
            //m *= m;
            //atemp = (asq - bsq) / m + x;
            //b = -(2 * a * b) / m + y;
            //a = atemp;

            e += 0.5 + 0.5 * sin(2.0 * atan2(b, a));
            //e += (asq + bsq - mn) / (Mn - mn);

            asq = a * a;
            bsq = b * b;
            ++k;
        }

        double lastOrbit = 0.5 + 0.5 * sin(2.0 * atan2(b, a));
        //double lastOrbit = (asq + bsq - mn) / (Mn - mn);

        double f = e - lastOrbit;
        e /= k;
        f /= k - 1;
        float frac = 1.0 + log(log(rsq) / log(asq + bsq)) / log(2.0);
        float mix = frac * e + (1 - frac) * f;
        float hue;

        //hue = mix;


        if (k == iterations) {
            hue = 1.0;
        } else if (mix >= 0) {
            hue = 1.0f * mix - (int)(1.0f * mix);
        } else {
            hue = 1.0f * mix - (int)(1.0f * mix) + 1;
        }

        //colour scheme
        //GLubyte colorArray[] = { 0, 0, 0, 255, 0, 0, 255, 155, 0, 255, 255, 255, 0, 0, 0 };
        //int length = 5;
        //GLubyte colorArray[] = { 0, 0, 0, 255, 100, 50, 0, 0, 0, 0, 100, 200, 50, 150, 255, 0, 0, 0 };
        //int length = 6;

        int n = (int)(hue * (colorSpectrumSize - 1));
        float h = hue * (colorSpectrumSize - 1) - n;

        GLubyte r1 = colorSpectrum[3 * n];
        GLubyte g1 = colorSpectrum[3 * n + 1];
        GLubyte b1 = colorSpectrum[3 * n + 2];
        GLubyte r2 = colorSpectrum[3 * n + 3];
        GLubyte g2 = colorSpectrum[3 * n + 4];
        GLubyte b2 = colorSpectrum[3 * n + 5];

        GLubyte R, G, B;
        R = r1 * (1 - h) + r2 * h;
        G = g1 * (1 - h) + g2 * h;
        B = b1 * (1 - h) + b2 * h;

        int j = 4 * i;
        data[j] = R;
        data[j + 1] = G;
        data[j + 2] = B;
        data[j + 3] = 255;
    }
}

void compute(GLubyte* data, const int width, const int height, const int iterations,
             const double midx, const double midy, const double scale,
             const double varx, const double vary, const bool julia,
             RenderType renderType, GLubyte* colorSpectrum, const int colorSpectrumSize)
{
    dim3 blockSize;
    blockSize.x = 16;
    blockSize.y = 16;

    dim3 gridSize;
    gridSize.x = width / blockSize.x + 1;
    gridSize.y = height / blockSize.y + 1;

    switch (renderType) {
        case Esc:
            computeEscape <<< gridSize, blockSize >>> (data, width, height, iterations, midx, midy, scale, varx, vary, julia, colorSpectrum, colorSpectrumSize);
            break;
        case Orbit:
            computeOrbit <<< gridSize, blockSize >>> (data, width, height, iterations, midx, midy, scale, 0, 0, julia, varx, vary, colorSpectrum, colorSpectrumSize);
            break;
        case Average:
            computeAverage <<< gridSize, blockSize >>> (data, width, height, iterations, midx, midy, scale, varx, vary, julia, colorSpectrum, colorSpectrumSize);
            break;
    }
}
